#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <chrono>
#include <mutex>

#include "constants.h"
#include "uv.h"
#include "messages.h"
#include "blake3.cu"
#include "pow.h"
#include "worker.h"
#include "template.h"
#include "mining.h"
#include "getopt.h"
#include "log.h"

std::atomic<uint32_t> found_solutions{0};

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::duration<double> duration_t;
typedef std::chrono::time_point<std::chrono::high_resolution_clock> time_point_t;

uv_loop_t *loop;
uv_stream_t *tcp;

time_point_t start_time = Time::now();

std::atomic<int> gpu_count;
std::atomic<int> worker_count;
std::atomic<uint64_t> total_mining_count;
std::atomic<uint64_t> device_mining_count[max_gpu_num];
bool use_device[max_gpu_num];

int port = 10973;
char broker_ip[16];
uv_timer_t reconnect_timer;
uv_tcp_t *uv_socket;
uv_connect_t *uv_connect;

void setup_gpu_worker_count(int _gpu_count, int _worker_count)
{
    gpu_count.store(_gpu_count);
    worker_count.store(_worker_count);
}

void on_write_end(uv_write_t *req, int status)
{
    if (status < 0)
    {
        LOGERR("error on_write_end %d\n", status);
    }
    free(req);
}

std::mutex write_mutex;
uint8_t write_buffer[4096 * 1024];
void submit_new_block(mining_worker_t *worker)
{
    expire_template_for_new_block(load_worker__template(worker));

    const std::lock_guard<std::mutex> lock(write_mutex);

    ssize_t buf_size = write_new_block(worker, write_buffer);
    uv_buf_t buf = uv_buf_init((char *)write_buffer, buf_size);
    print_hex("new solution", (uint8_t *) hasher_buf(worker, true), 32);

    uv_write_t *write_req = (uv_write_t *)malloc(sizeof(uv_write_t));
    uint32_t buf_count = 1;

    uv_write(write_req, tcp, &buf, buf_count, on_write_end);
    found_solutions.fetch_add(1, std::memory_order_relaxed);
}

void mine_with_timer(uv_timer_t *timer);

void mine(mining_worker_t *worker)
{
    time_point_t start = Time::now();

    int32_t to_mine_index = next_chain_to_mine();
    if (to_mine_index == -1)
    {
        LOG("waiting for new tasks\n");
        worker->timer.data = worker;
        uv_timer_start(&worker->timer, mine_with_timer, 500, 0);
    } else {
        mining_counts[to_mine_index].fetch_add(mining_steps);
        setup_template(worker, load_template(to_mine_index));

        start_worker_mining(worker);

        duration_t elapsed = Time::now() - start;
        // LOG("=== mining time: %fs\n", elapsed.count());
    }
}

void mine_with_req(uv_work_t *req)
{
    mining_worker_t *worker = load_req_worker(req);
    mine(worker);
}

void mine_with_async(uv_async_t *handle)
{
    mining_worker_t *worker = (mining_worker_t *)handle->data;
    mine(worker);
}

void mine_with_timer(uv_timer_t *timer)
{
    mining_worker_t *worker = (mining_worker_t *)timer->data;
    mine(worker);
}

void after_mine(uv_work_t *req, int status)
{
    return;
}

void worker_stream_callback(hipStream_t stream, hipError_t status, void *data)
{
    mining_worker_t *worker = (mining_worker_t *)data;
    if (hasher_found_good_hash(worker, true))
    {
        store_worker_found_good_hash(worker, true);
        submit_new_block(worker);
    }

    mining_template_t *template_ptr = load_worker__template(worker);
    job_t *job = template_ptr->job;
    uint32_t chain_index = job->from_group * group_nums + job->to_group;
    mining_counts[chain_index].fetch_sub(mining_steps);
    mining_counts[chain_index].fetch_add(hasher_hash_count(worker, true));
    total_mining_count.fetch_add(hasher_hash_count(worker, true));
    device_mining_count[worker->device_id].fetch_add(hasher_hash_count(worker, true));
    free_template(template_ptr);
    worker->async.data = worker;
    uv_async_send(&(worker->async));
}

void start_mining()
{
    assert(mining_templates_initialized == true);

    start_time = Time::now();

    for (uint32_t i = 0; i < worker_count.load(); i++)
    {
        if (use_device[mining_workers[i].device_id])
        {
            uv_queue_work(loop, &req[i], mine_with_req, after_mine);
        }
    }
}

void start_mining_if_needed()
{
    if (!mining_templates_initialized)
    {
        bool all_initialized = true;
        for (int i = 0; i < chain_nums; i++)
        {
            if (load_template(i) == NULL)
            {
                all_initialized = false;
                break;
            }
        }
        if (all_initialized)
        {
            mining_templates_initialized = true;
            start_mining();
        }
    }
}

void alloc_buffer(uv_handle_t *handle, size_t suggested_size, uv_buf_t *buf)
{
    buf->base = (char *)malloc(suggested_size);
    buf->len = suggested_size;
}

void log_hashrate(uv_timer_t *timer)
{
    time_point_t current_time = Time::now();
    if (current_time > start_time)
    {
        duration_t eplased = current_time - start_time;
        LOG("hashrate: %.0f MH/s ", total_mining_count.load() / eplased.count() / 1000000);
        for (int i = 0; i < gpu_count; i++)
        {
            LOG_WITHOUT_TS("gpu%d: %.0f MH/s ", i, device_mining_count[i].load() / eplased.count() / 1000000);
        }
        LOG_WITHOUT_TS("solutions: %u\n", found_solutions.load(std::memory_order_relaxed));
    }
}

uint8_t read_buf[2048 * 1024 * chain_nums];
blob_t read_blob = {read_buf, 0};
server_message_t *decode_buf(const uv_buf_t *buf, ssize_t nread)
{
    if (read_blob.len == 0)
    {
        read_blob.blob = (uint8_t *)buf->base;
        read_blob.len = nread;
        server_message_t *message = decode_server_message(&read_blob);
        if (message)
        {
            // some bytes left
            if (read_blob.len > 0)
            {
                memcpy(read_buf, read_blob.blob, read_blob.len);
                read_blob.blob = read_buf;
            }
            return message;
        }
        else
        { // no bytes consumed
            memcpy(read_buf, buf->base, nread);
            read_blob.blob = read_buf;
            read_blob.len = nread;
            return NULL;
        }
    }
    else
    {
        assert(read_blob.blob == read_buf);
        memcpy(read_buf + read_blob.len, buf->base, nread);
        read_blob.len += nread;
        return decode_server_message(&read_blob);
    }
}

void connect_to_broker();

void try_to_reconnect(uv_timer_t *timer){
    read_blob.len = 0;
    free(uv_socket);
    free(uv_connect);
    connect_to_broker();
    uv_timer_stop(timer);
}

void on_read(uv_stream_t *server, ssize_t nread, const uv_buf_t *buf)
{
    if (nread < 0)
    {
        LOGERR("error on_read %ld: might be that the full node is not synced, or miner wallets are not setup, try to reconnect\n", nread);
        uv_timer_start(&reconnect_timer, try_to_reconnect, 5000, 0);
        return;
    }

    if (nread == 0)
    {
        return;
    }

    server_message_t *message = decode_buf(buf, nread);
    if (message)
    {
        switch (message->kind)
        {
        case JOBS:
            for (int i = 0; i < message->jobs->len; i++)
            {
                update_templates(message->jobs->jobs[i]);
            }
            start_mining_if_needed();
            break;

        case SUBMIT_RESULT:
            LOG("submitted: %d -> %d: %d \n", message->submit_result->from_group, message->submit_result->to_group, message->submit_result->status);
            break;
        }
        free_server_message_except_jobs(message);
    }

    free(buf->base);
    // uv_close((uv_handle_t *) server, free_close_cb);
}

void on_connect(uv_connect_t *req, int status)
{
    if (status < 0)
    {
        LOGERR("connection error %d: might be that the full node is not reachable, try to reconnect\n", status);
        uv_timer_start(&reconnect_timer, try_to_reconnect, 5000, 0);
        return;
    }
    LOG("the server is connected %d %p\n", status, req);

    tcp = req->handle;
    uv_read_start(req->handle, alloc_buffer, on_read);
}

void connect_to_broker(){
    uv_socket = (uv_tcp_t *)malloc(sizeof(uv_tcp_t));
    uv_tcp_init(loop, uv_socket);
    uv_tcp_nodelay(uv_socket, 1);
    uv_connect = (uv_connect_t *)malloc(sizeof(uv_connect_t));
    struct sockaddr_in dest;
    uv_ip4_addr(broker_ip, port, &dest);
    uv_tcp_connect(uv_connect, uv_socket, (const struct sockaddr *)&dest, on_connect);
}

bool is_valid_ip_address(char *ip_address)
{
    struct sockaddr_in sa;
    int result = inet_pton(AF_INET, ip_address, &(sa.sin_addr));
    return result != 0;
}

int hostname_to_ip(char *ip_address, char *hostname)
{
    struct addrinfo hints, *servinfo;
    memset(&hints, 0, sizeof hints);
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_STREAM;

    int res = getaddrinfo(hostname, NULL, &hints, &servinfo);
    if (res != 0)
    {
        LOGERR("getaddrinfo: %s\n", gai_strerror(res));
        return 1;
    }

    struct sockaddr_in *h = (struct sockaddr_in *)servinfo->ai_addr;
    strcpy(ip_address, inet_ntoa(h->sin_addr));

    freeaddrinfo(servinfo);
    return 0;
}
#ifndef MINER_VERSION
#define MINER_VERSION "unknown"
#endif

int main(int argc, char **argv)
{
    setbuf(stdout, NULL);

    #ifdef _WIN32
    WSADATA wsa;
    // current winsocket version is 2.2
    int rc = WSAStartup(MAKEWORD(2, 2), &wsa);
    if (rc != 0)
    {
        LOGERR("Initialize winsock failed: %d\n", rc);
        exit(1);
    }
    #endif

    LOG("Running gpu-miner version : %s\n", MINER_VERSION);

    int gpu_count = 0;
    hipGetDeviceCount(&gpu_count);
    LOG("GPU count: %d\n", gpu_count);
    for (int i = 0; i < gpu_count; i++)
    {
        LOG("GPU #%d has #%d cores\n", i, get_device_cores(i));
        use_device[i] = true;
    }

    strcpy(broker_ip, "127.0.0.1");

    int command;
    while ((command = getopt(argc, argv, "p:g:a:")) != -1)
    {
        switch (command)
        {
        case 'p':
            port = atoi(optarg);
            break;
        case 'a':
            if (is_valid_ip_address(optarg))
            {
                strcpy(broker_ip, optarg);
            }
            else
            {
                hostname_to_ip(broker_ip, optarg);
            }
            break;

        case 'g':
            for (int i = 0; i < gpu_count; i++)
            {
                use_device[i] = false;
            }
            optind--;
            for (; optind < argc && *argv[optind] != '-'; optind++)
            {
                int device = atoi(argv[optind]);
                if (device < 0 || device >= gpu_count) {
                    LOGERR("Invalid gpu index %d\n", device);
                    exit(1);
                }
                use_device[device] = true;
            }
            break;
        default:
            LOGERR("Invalid command %c\n", command);
            exit(1);
        }
    }
    LOG("will connect to broker @%s:%d\n", broker_ip, port);

    #ifdef __linux__
    signal(SIGPIPE, SIG_IGN);
    #endif

    mining_workers_init(gpu_count);
    setup_gpu_worker_count(gpu_count, gpu_count * parallel_mining_works_per_gpu);

    loop = uv_default_loop();
    uv_timer_init(loop, &reconnect_timer);
    connect_to_broker();

    for (int i = 0; i < worker_count; i++)
    {
        uv_async_init(loop, &(mining_workers[i].async), mine_with_async);
        uv_timer_init(loop, &(mining_workers[i].timer));
    }

    uv_timer_t log_timer;
    uv_timer_init(loop, &log_timer);
    uv_timer_start(&log_timer, log_hashrate, 5000, 20000);

    uv_run(loop, UV_RUN_DEFAULT);

    return (0);
}
